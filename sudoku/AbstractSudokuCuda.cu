
#include <thread>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "AbstractSudokuCuda.h"

// hack
#define __globals__

AbstractSudokuCuda::AbstractSudokuCuda(const char initialMatrix[LENGTH][LENGTH])
{
    hipMalloc((void **)&this->matrix, sizeof(MatrixCuda));
    *matrix = MatrixCuda(initialMatrix);
}

void
AbstractSudokuCuda::solve()
{
    while (!this->matrix->solved())
    {
        this->solveInRange(1, 9);
    }
}

void AbstractSudokuCuda::print()
{
    this->matrix->print();
}

__globals__ void solveInRangeCuda(MatrixCuda *matrix, 
                                 MatrixCuda *currentMatrix, 
                                 int number)
{
    matrix->findSolutionForANumber(number);
    currentMatrix->merge(*matrix);
}

void
AbstractSudokuCuda::solveInRange(int from, int to)
{
    int number = from;

    do
    {
        MatrixCuda *newMatrix = nullptr;
        hipMalloc((void **)newMatrix, sizeof(MatrixCuda));
        *newMatrix = MatrixCuda(*matrix);
        solveInRangeCuda(newMatrix, matrix, number);
        number++;
    } while (number <= to);
}

AbstractSudokuCuda::~AbstractSudokuCuda()
{
    hipFree(this->matrix);
}